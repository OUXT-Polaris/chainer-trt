#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018 Preferred Networks, Inc. All rights reserved.
 */

#include <hip/hip_fp16.h>

namespace chainer_trt {
namespace plugin {
    template <typename T, unsigned int KH, unsigned int KW>
    __global__ void shift_kernel(const T* __restrict__ x, const int c,
                                 const int h, const int dy, const int dx,
                                 T* __restrict__ y) {
        const unsigned int w = blockDim.x + 1024 * (gridDim.x - 1);
        const unsigned int hw = h * w;
        // blockIdx.z is batch index
        const unsigned int batch_offset = blockIdx.z * c * hw;

        int kx = 0;
        int ky = 0;
        {
            const unsigned int n_groups = KH * KW;
            const unsigned int group_size = c / n_groups;

            // blockIdx.y is channel index
            unsigned int group_idx = blockIdx.y / group_size;

            // Make sure that center group is last
            if(group_idx == (n_groups - 1) / 2)
                group_idx = n_groups - 1;
            else if(group_idx == n_groups - 1)
                group_idx = (n_groups - 1) / 2;

            if(group_idx < n_groups) {
                ky = (group_idx / KW) - KH / 2;
                kx = (group_idx % KW) - KW / 2;
            }
        }

        unsigned int offset = batch_offset + blockIdx.y * hw;
        // blockIdx.x is width index
        const int out_col = threadIdx.x + 1024 * blockIdx.x;
        int out_row = 0;

        y = &y[offset + out_col];

        int in_col = -kx * dx + out_col;
        if(in_col >= 0 && in_col < w) {
            offset += in_col;
            int in_row_offset = ky * dy;

            for(; out_row < in_row_offset; out_row++) {
                *y = 0;
                y += w;
            }

            unsigned int copy_h = min(h, h + in_row_offset);
            for(; out_row < copy_h; out_row++) {
                int in_row = -in_row_offset + out_row;
                *y = x[offset + in_row * w];
                y += w;
            }
        }
        for(; out_row < h; out_row++) {
            *y = 0;
            y += w;
        }
    }

    template <typename T>
    __global__ void
    shift_fallback_kernel(const T* __restrict__ x, const int c, const int h,
                          const int w, const int kh, const int kw, const int dy,
                          const int dx, T* __restrict__ y) {
        const unsigned int hw = h * w;
        const unsigned int chw = c * hw;
        // blockDim.y is batch index
        const unsigned int ofst = blockIdx.y * chw;
        const unsigned int n_groups = kh * kw;
        const unsigned int group_size = c / n_groups;
        const unsigned int stride = blockDim.x * gridDim.x;

#pragma unroll
        for(unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < chw;
            i += stride) {
            // Based on
            // https://github.com/chainer/chainer/pull/4041/files#diff-4cb6895e9a4acd09178c617ff177d0d0
            unsigned int b0 = i / chw;
            unsigned int rest = (i % chw);
            unsigned int c0 = rest / hw;
            rest %= hw;

            unsigned int out_row = rest / w;
            unsigned int out_col = rest % w;

            unsigned int group_idx = c0 / group_size;

            if(group_idx == (n_groups - 1) / 2)
                group_idx = n_groups - 1;
            else if(group_idx == n_groups - 1)
                group_idx = (n_groups - 1) / 2;

            unsigned int ky = (group_idx / kw) - kh / 2;
            unsigned int kx = (group_idx % kw) - kw / 2;
            if(group_idx >= n_groups) {
                ky = 0;
                kx = 0;
            }

            signed int in_row = -ky * dy + out_row;
            signed int in_col = -kx * dx + out_col;
            if(in_row >= 0 && in_row < h && in_col >= 0 && in_col < w)
                y[ofst + i] =
                  x[ofst + b0 * chw + c0 * hw + in_row * w + in_col];
            else
                y[ofst + i] = 0;
        }
    }

    template <typename T>
    void apply_shift(const T* src_gpu, int batch_size, int c, int h, int w,
                     int kh, int kw, int dy, int dx, int grid_size,
                     int block_size, T* dst_gpu, hipStream_t stream) {
        if(w < 1024 || w % 1024 == 0) {
            dim3 grid(max(1, w / 1024), c, batch_size);
            if(kh == 7 && kw == 7) {
                shift_kernel<T, 7, 7><<<grid, min(w, 1024), 0, stream>>>(
                  src_gpu, c, h, dy, dx, dst_gpu);
                return;
            } else if(kh == 5 && kw == 5) {
                shift_kernel<T, 5, 5><<<grid, min(w, 1024), 0, stream>>>(
                  src_gpu, c, h, dy, dx, dst_gpu);
                return;
            } else if(kh == 3 && kw == 3) {
                shift_kernel<T, 3, 3><<<grid, min(w, 1024), 0, stream>>>(
                  src_gpu, c, h, dy, dx, dst_gpu);
                return;
            }
        }

        dim3 grid(grid_size, batch_size);
        shift_fallback_kernel<T><<<grid, block_size, 0, stream>>>(
          src_gpu, c, h, w, kh, kw, dy, dx, dst_gpu);
    }

    template void apply_shift(const float*, int, int, int, int, int, int, int,
                              int, int, int, float*, hipStream_t);
    template void apply_shift(const __half*, int, int, int, int, int, int, int,
                              int, int, int, __half*, hipStream_t);
}
}
