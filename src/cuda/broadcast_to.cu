/*
 * Copyright (c) 2018 Preferred Networks, Inc. All rights reserved.
 */


#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

namespace chainer_trt {
namespace plugin {
    template <typename T>
    __global__ void broadcast_to_kernel(const T* d_src, T* d_dst,
                                        int* d_i_strides, int* d_o_strides,
                                        int in_size, int out_size,
                                        int nb_dims) {
        const int idx = blockIdx.x * blockDim.x + threadIdx.x;
        if(idx < out_size) {
            // calc offset relationship between input & output
            int in_idx = 0;
            int f = idx;
            for(int i = 0; i < nb_dims; i++) {
                in_idx += (f / d_o_strides[i]) * d_i_strides[i];
                f = f % d_o_strides[i];
            }
            d_dst[blockIdx.y * out_size + idx] =
              d_src[blockIdx.y * in_size + in_idx];
        }
    }

    template <typename T>
    void apply_broadcast_to(const T* d_src, T* d_dst, int* d_i_strides,
                            int* d_o_strides, int in_size, int out_size,
                            int nb_dims, int batch_size, hipStream_t stream) {
        const int thread_size = 1024;
        const int block_size = (int)std::ceil(1.0 * out_size / thread_size);
        dim3 grid(block_size, batch_size);
        broadcast_to_kernel<<<grid, thread_size, 0, stream>>>(
          d_src, d_dst, d_i_strides, d_o_strides, in_size, out_size, nb_dims);
    }

    template void apply_broadcast_to(const float*, float*, int*, int*, int, int,
                                     int, int, hipStream_t);
    template void apply_broadcast_to(const __half*, __half*, int*, int*, int,
                                     int, int, int, hipStream_t);
}
}
